
#include <hip/hip_runtime.h>
__global__
void myKernel(int *in) {
    in[threadIdx.x] += 1;
}

int main(void) {
    int *dIn;
    hipMalloc(&dIn, sizeof(int));

    myKernel<<<1, 128>>>(dIn);
    return 0;
}
