#include "hip/hip_runtime.h"
#include "../utils.h"

__global__
void myKernel(int *in) {
    in[threadIdx.x] += 1;
}

int main(void) {
    int *dIn;
    checkCudaErrors(hipMalloc(&dIn, sizeof(int)));

    myKernel<<<1, 2>>>(dIn);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    return 0;
}
