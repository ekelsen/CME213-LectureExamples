#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include "../utils.h"

__global__
void kernel(int *out) {
  out[threadIdx.x] = threadIdx.x;
}

int main(int argc, char **argv) {
  int N = 32;

  if (argc == 2)
    N = atoi(argv[1]);

  int *d_output;

  std::vector<int> h_output(N);

  checkCudaErrors(hipMalloc(&d_output, sizeof(int) * N));

  kernel<<<1, N>>>(d_output);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(&h_output[0], d_output, sizeof(int) * N, hipMemcpyDeviceToHost));

  for (int i = 0; i < N; ++i)
    std::cout << h_output[i] << std::endl;

  checkCudaErrors(hipFree(d_output));

  return 0;
}
