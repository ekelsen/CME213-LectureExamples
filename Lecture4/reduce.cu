#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <limits>
#include "../utils.h"

__global__
void reduceKernel(const float* const input, float *sum, int N)
{
  const int lane = threadIdx.x;
  const int warp = threadIdx.y;

  __shared__ float smem[32];

  float myVal = input[lane];

  smem[lane] = myVal;

  __syncthreads();

  for (int shift = 16; shift > 0; shift >>= 1) {
    if (lane < shift) {
      smem[lane] += smem[lane + shift];
    }
    __syncthreads();
  }

  if (lane == 0)
    *sum = smem[lane];
}

int main(void) {
  const int N = 32;

  std::vector<float> h_input(N);

  float h_sum = 0.f;

  for (int i = 0; i < N; ++i) {
    h_input[i] = (rand() / (double)std::numeric_limits<int>::max());
    h_sum += h_input[i];
  }

  float *d_input;
  checkCudaErrors(hipMalloc(&d_input, N * sizeof(int)));
  checkCudaErrors(hipMemcpy(d_input, &h_input[0], N * sizeof(int), hipMemcpyHostToDevice));

  float *d_sum;
  checkCudaErrors(hipMalloc(&d_sum, sizeof(int)));

  reduceKernel<<<1, 32>>>(d_input, d_sum, N);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  float h_d_sum;
  checkCudaErrors(hipMemcpy(&h_d_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost));

  std::cout << "cpu: " << h_sum << " gpu: " << h_d_sum << std::endl;


  return 0;
}
