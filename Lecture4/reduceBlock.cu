#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <limits>
#include "../utils.h"

template<int blockSize>
__global__
void reduceWarp(const int* const input, int *sum)
{
  const int tid = threadIdx.x;

  __shared__ int smem[blockSize];

  int myVal = input[tid];

  smem[tid] = myVal;

  __syncthreads();

  //use this for non-power of 2 blockSizes
  //for (int shift = (1 << (32 - __clz(blockSize - 1))); shift > 0; shift >>= 1) {
  //  if (tid + shift < blockSize) {
  for (int shift = blockSize / 2; shift > 0; shift >>= 1) {
    if (tid < shift) {
      smem[tid] += smem[tid + shift];
    }
    __syncthreads();
  }

  if (tid == 0)
    *sum = smem[tid];
}

int main(void) {
  const int N = 128;

  std::vector<int> h_input(N);

  int h_sum = 0.f;

  for (int i = 0; i < N; ++i) {
    h_input[i] = rand() % 10;
    h_sum += h_input[i];
  }

  int *d_input;
  checkCudaErrors(hipMalloc(&d_input, N * sizeof(int)));
  checkCudaErrors(hipMemcpy(d_input, &h_input[0], N * sizeof(int), hipMemcpyHostToDevice));

  int *d_sum;
  checkCudaErrors(hipMalloc(&d_sum, sizeof(int)));

  reduceWarp<N><<<1, N>>>(d_input, d_sum);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  int h_d_sum;
  checkCudaErrors(hipMemcpy(&h_d_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost));

  std::cout << "cpu: " << h_sum << " gpu: " << h_d_sum << std::endl;


  return 0;
}
